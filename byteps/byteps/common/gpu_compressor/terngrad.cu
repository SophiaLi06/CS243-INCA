#include "hip/hip_runtime.h"
#include "terngrad.cuh"
#include "math.h"
#include <stdio.h>
#include <iostream>

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void find_grad_max(const void* gpu_ptr, size_t len, float* result){
    float* ptr = reinterpret_cast<float*>(const_cast<void*>(gpu_ptr));
    // printf("find_grad_max sample: %p, %.6f, %.6f, %.6f, %.6f, %.6f \n", gpu_ptr,
    //        ptr[0], ptr[1], ptr[2], ptr[3], ptr[4]);

    float grad_max;
    if (ptr[0] >= 0) grad_max = ptr[0];
    else grad_max = -ptr[0];
    float grad_abs;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(size_t i = index; i < len; i+=stride){
        if (ptr[i] >= 0) grad_abs = ptr[i];
        else grad_abs = -ptr[i];
        if (grad_abs > grad_max) grad_max = grad_abs;
    }
    *result = grad_max;
}

__global__ void para_max(const void* gpu_ptr, size_t len, float* result){
    extern __shared__ float res_cache[];

    float* ptr = reinterpret_cast<float*>(const_cast<void*>(gpu_ptr));

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int cacheIndex = threadIdx.x;

    float res = -1.0;
    for(size_t i = index; i < len; i+=stride){
        if (fabsf(ptr[i]) > res) res = fabsf(ptr[i]);
    }

    res_cache[cacheIndex] = res; // set the result cache value
    __syncthreads();

    // Perform parallel reduction
    int inc = blockDim.x / 2;
    while (inc != 0){
        if (cacheIndex < inc && res_cache[cacheIndex + inc] > res_cache[cacheIndex]) {
            res_cache[cacheIndex] = res_cache[cacheIndex + inc];
        }

        __syncthreads();
        inc /= 2;
    }

    if (cacheIndex == 0) result[blockIdx.x] = res_cache[0];
}

__global__ void terngrad_compress_kernel(const void* gpu_ptr, size_t len, hiprandState *state, float grad_max){
    //threadIdx.x contains the index of the current thread within its block, 
    //and blockDim.x contains the number of threads in the block
    //and gridDim.x gives the number of blocks in a grid
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    float* ptr = reinterpret_cast<float*>(const_cast<void*>(gpu_ptr));
    
    // if (id == 0) {
    //     printf("compress sample: %p, %.6f, %.6f, %.6f, %.6f, %.6f \n", gpu_ptr,
    //            ptr[0], ptr[1], ptr[2], ptr[3], ptr[4]);
    // }
    float x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    for(size_t i = index; i < len; i+=stride) {
        x = hiprand_uniform(&localState);
        if(x < fabsf(ptr[i])/grad_max) {
            if (ptr[i] > 0) ptr[i] = 1.0;
            else ptr[i] = -1.0;
        }
        else ptr[i] = 0.0;
    }
    // /* use the last value of gpu_ptr to store the "scaling factor" grad_max */
    // ptr[len-1] = grad_max;
    /* Copy state back to global memory */
    state[id] = localState;
    // TODO: change data type from float to uint8
}

__global__ void terngrad_decompress_kernel(const void* gpu_ptr, size_t len, float scale){
    //threadIdx.x contains the index of the current thread within its block, 
    //and blockDim.x contains the number of threads in the block
    //and gridDim.x gives the number of blocks in a grid
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    float* ptr = reinterpret_cast<float*>(const_cast<void*>(gpu_ptr));
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(size_t i = index; i < len-1; i+=stride) {
        ptr[i] *= scale;
    }
}

float terngrad_scale(const void* gpu_ptr, size_t len){
    float* ptr = reinterpret_cast<float*>(const_cast<void*>(gpu_ptr));
    
    float grad_max = 0.0;

    if (len > 200){
        float *host_max_res, *dev_max_res;

        const unsigned int maxBlockCount = 32;
        const unsigned int maxThreadPerBlock = 128;
    
        // Allocate space for result on host
        host_max_res = (float*)calloc(maxBlockCount, sizeof(float));
        // Allocate space for result on device
        hipMalloc(&dev_max_res, maxBlockCount * sizeof(float));
        para_max<<<maxBlockCount, maxThreadPerBlock, maxThreadPerBlock * sizeof(float)>>>(gpu_ptr, len, dev_max_res);

        // Copy device result to host
        hipMemcpy(host_max_res, dev_max_res, maxBlockCount * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(dev_max_res);

        // Find the maximum value across all blocks
        for (int i = 0; i < maxBlockCount; ++i){
            if (host_max_res[i] > grad_max) grad_max = host_max_res[i];
        }
    }
    else{
        float* grad_max_answer;
        hipMalloc(&grad_max_answer, sizeof(float));
        //find_grad_max<<<64, 64>>>(gpu_ptr, len, grad_max_answer);
        find_grad_max<<<1, 1>>>(gpu_ptr, len, grad_max_answer);
        hipMemcpy(&grad_max, grad_max_answer, sizeof(float), hipMemcpyDeviceToHost);
        hipFree(grad_max_answer);
    }
    return grad_max;
}

float terngrad_compress(const void* gpu_ptr, size_t len, float scale){
    // This is a tensor used for push-pulling the accuracy
    // TODO: in the future. just don't call terngrad_compress and decompress on it
    if (len <= 2) return 1.0; 
#ifdef TOTAL_TIME_CUDA
    // Create the timer
    hipEvent_t total_start, total_stop;
    hipEventCreate(&total_start);
    hipEventCreate(&total_stop);

    // Start the timer
    hipEventRecord(total_start, 0);
#endif
    float* ptr = reinterpret_cast<float*>(const_cast<void*>(gpu_ptr));

    // std::cout << "compress sample: " << gpu_ptr << " " << ptr[0]
    //           << " " << ptr[1] << " " << ptr[2] << " " << ptr[3] << " " 
    //           << ptr[4] << " " << ptr[5] << std::endl;
    
    //float grad_max = 0.0;

#ifdef TIME_CUDA
    // Create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start the timer
    hipEventRecord(start, 0);
#endif
    // if (len > 200){
    //     float *host_max_res, *dev_max_res;

    //     const unsigned int maxBlockCount = 32;
    //     const unsigned int maxThreadPerBlock = 128;
    
    //     // Allocate space for result on host
    //     host_max_res = (float*)calloc(maxBlockCount, sizeof(float));
    //     // Allocate space for result on device
    //     hipMalloc(&dev_max_res, maxBlockCount * sizeof(float));
    //     para_max<<<maxBlockCount, maxThreadPerBlock, maxThreadPerBlock * sizeof(float)>>>(gpu_ptr, len, dev_max_res);

    //     // Copy device result to host
    //     hipMemcpy(host_max_res, dev_max_res, maxBlockCount * sizeof(float), hipMemcpyDeviceToHost);
    //     hipFree(dev_max_res);

    //     // Find the maximum value across all blocks
    //     for (int i = 0; i < maxBlockCount; ++i){
    //         if (host_max_res[i] > grad_max) grad_max = host_max_res[i];
    //     }
    // }
    // else{
    //     float* grad_max_answer;
    //     hipMalloc(&grad_max_answer, sizeof(float));
    //     //find_grad_max<<<64, 64>>>(gpu_ptr, len, grad_max_answer);
    //     find_grad_max<<<1, 1>>>(gpu_ptr, len, grad_max_answer);
    //     hipMemcpy(&grad_max, grad_max_answer, sizeof(float), hipMemcpyDeviceToHost);
    //     hipFree(grad_max_answer);
    // }

#ifdef TIME_CUDA
    // Stop the timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float find_max_time;
    hipEventElapsedTime(&find_max_time, start, stop);
    std::cout << "Time to find grad_max: " << find_max_time << std::endl;
#endif

    const unsigned int threadsPerBlock = 512;
    //const unsigned int blockCount = 64;
    const unsigned int blockCount = (len + threadsPerBlock - 1) / threadsPerBlock;
    const unsigned int totalThreads = threadsPerBlock * blockCount;
    hiprandState *devStates;

    // /* Allocate space for results on host */
    // hostResults = (unsigned int *)calloc(totalThreads, sizeof(int));

    /* Allocate space for prng states on device */
    hipMalloc((void**)&devStates, totalThreads * sizeof(hiprandState));
    //std::cout << "Done mallocing for devStates" << std::endl;

#ifdef TIME_CUDA
    // Start the timer
    hipEventRecord(start, 0);
#endif
    /* Setup prng states */
    setup_kernel<<<blockCount, threadsPerBlock>>>(devStates);

    // scale is zero, should not divide by grad_max, just return for now
    // TODO: should investigate how to avoid compressing/decompressing parameter tensors
    if(scale) terngrad_compress_kernel<<<blockCount, threadsPerBlock>>>(gpu_ptr, len, devStates, scale);
    //terngrad_compress_kernel<<<blockCount, threadsPerBlock>>>(gpu_ptr, len, devStates, grad_max);
    //if(scale != grad_max) std::cout << "grad max: "<< grad_max << " scale: " << scale;
    //std::cout << "grad max: "<< grad_max << " host scale: " << scale;

#ifdef TIME_CUDA
    // Stop the timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float find_terngrad_time;
    hipEventElapsedTime(&find_terngrad_time, start, stop);
    std::cout << "Time to compress w/ terngrad: " << find_terngrad_time << std::endl;
#endif
#ifdef TOTAL_TIME_CUDA
    // Stop the timer
    hipEventRecord(total_stop, 0);
    hipEventSynchronize(total_stop);
    float total_terngrad_time;
    hipEventElapsedTime(&total_terngrad_time, total_start, total_stop);
    std::cout << "Total time to compress w/ terngrad: " << total_terngrad_time << std::endl;
#endif
    /* Cleanup */
    hipFree(devStates);
#ifdef TIME_CUDA
    hipEventDestroy(start);
    hipEventDestroy(stop);
#endif
#ifdef TOTAL_TIME_CUDA
    hipEventDestroy(total_start);
    hipEventDestroy(total_stop);
#endif
    return scale;
}

void terngrad_decompress(const void* gpu_ptr, float scale, size_t len){
    // This is a tensor used for push-pulling the accuracy
    // TODO: in the future. just don't call terngrad_compress and decompress on it
    if (len <= 2) return;

    // maybe we can just "throw away" the last floating point value of the old
    // gradient (assume it's zero) and use that space to store the scale
    // Later we can just play with the size of the tensor when we initailze them (shrink for compression
    // + 1 for storing the scale)
    const unsigned int threadsPerBlock = 512;
    //const unsigned int blockCount = 64;
    const unsigned int blockCount = (len + threadsPerBlock - 1) / threadsPerBlock;
    if(scale) terngrad_decompress_kernel<<<blockCount, threadsPerBlock>>>(gpu_ptr, len, scale);
    return;
}
